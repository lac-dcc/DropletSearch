#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 96;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 165, KH = 7;
int NW = 165, KW = 7;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(480) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ kernel) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[5130];
  __shared__ float compute_shared[49];
  float PaddedInput_shared_local[2];
  float compute_shared_local[1];
  DepthwiseConv2d_local[0] = 0.000000e+00f;
  DepthwiseConv2d_local[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = ((((3 <= (((((int)blockIdx.x) % 17) * 10) + (((int)threadIdx.x) / 171))) && (3 <= (((int)threadIdx.x) % 171))) && ((((int)threadIdx.x) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + ((((int)threadIdx.x) / 171) * 165)) + (((int)threadIdx.x) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 480)] = ((((3 <= (((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 480) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 46) % 57))) && (((((int)threadIdx.x) + 138) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 480) / 171) * 165)) + ((((int)threadIdx.x) + 138) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 960)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 960) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 35) % 57))) && (((((int)threadIdx.x) + 105) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 960) / 171) * 165)) + ((((int)threadIdx.x) + 105) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 1440) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1440)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 1440) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 24) % 57))) && (((((int)threadIdx.x) + 72) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 1440) / 171) * 165)) + ((((int)threadIdx.x) + 72) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 1920) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1920)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 1920) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 13) % 57))) && (((((int)threadIdx.x) + 39) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 1920) / 171) * 165)) + ((((int)threadIdx.x) + 39) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 17) * 10) + ((((((int)threadIdx.x) / 3) + 800) % 855) / 57)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 2400)] = (((((3 <= (((((int)blockIdx.x) % 17) * 10) + ((((((int)threadIdx.x) / 3) + 800) % 855) / 57))) && ((((((int)blockIdx.x) % 17) * 5) + ((((((int)threadIdx.x) / 3) + 800) % 855) / 114)) < 84)) && (1 <= (((((int)threadIdx.x) / 3) + 2) % 57))) && (((((int)threadIdx.x) + 6) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 2400) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((((int)threadIdx.x) / 3) + 800) % 855) / 57) * 165)) + ((((int)threadIdx.x) + 6) % 171)) - 498)] : 0.000000e+00f);
  }
  PaddedInput_shared[(((int)threadIdx.x) + 2880)] = ((((3 <= (((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 315) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 48) % 57))) && (((((int)threadIdx.x) + 144) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 2880) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 315) / 171) * 165)) + ((((int)threadIdx.x) + 144) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3360)] = (((1 <= (((((int)threadIdx.x) / 3) + 37) % 57)) && (((((int)threadIdx.x) + 111) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 3360) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 795) / 171) * 165)) + ((((int)threadIdx.x) + 111) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3840)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 1275) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 26) % 57))) && (((((int)threadIdx.x) + 78) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 3840) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 1275) / 171) * 165)) + ((((int)threadIdx.x) + 78) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 1755) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 4320)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 1755) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 15) % 57))) && (((((int)threadIdx.x) + 45) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 4320) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 1755) / 171) * 165)) + ((((int)threadIdx.x) + 45) % 171)) - 498)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 330) {
    if ((((((int)blockIdx.x) % 17) * 10) + ((((int)threadIdx.x) + 2235) / 171)) < 171) {
      PaddedInput_shared[(((int)threadIdx.x) + 4800)] = (((((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) + 2235) / 342)) < 84) && (1 <= (((((int)threadIdx.x) / 3) + 4) % 57))) && (((((int)threadIdx.x) + 12) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 1632) * 5227200) + (((((int)threadIdx.x) + 4800) / 2565) * 2613600)) + (((((int)blockIdx.x) % 1632) / 17) * 27225)) + ((((int)blockIdx.x) % 17) * 1650)) + (((((int)threadIdx.x) + 2235) / 171) * 165)) + ((((int)threadIdx.x) + 12) % 171)) - 498)] : 0.000000e+00f);
    }
  }
  if (((int)threadIdx.x) < 49) {
    compute_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) % 1632) / 17) * 49) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner_outer = 0; k_inner_outer < 49; ++k_inner_outer) {
    if (((((((int)blockIdx.x) % 17) * 10) + (((((int)threadIdx.x) % 240) / 48) * 2)) + (k_inner_outer / 7)) < 171) {
      PaddedInput_shared_local[0] = PaddedInput_shared[((((((((int)threadIdx.x) / 240) * 2565) + (((((int)threadIdx.x) % 240) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7))];
      if ((((((int)threadIdx.x) % 48) * 2) + (k_inner_outer % 7)) < 75) {
        PaddedInput_shared_local[1] = PaddedInput_shared[(((((((((int)threadIdx.x) / 240) * 2565) + (((((int)threadIdx.x) % 240) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7)) + 96)];
      }
    }
    compute_shared_local[0] = compute_shared[k_inner_outer];
    if ((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) % 240) / 48)) < 83) {
      DepthwiseConv2d_local[0] = (DepthwiseConv2d_local[0] + (PaddedInput_shared_local[0] * compute_shared_local[0]));
      if ((((int)threadIdx.x) % 48) < 35) {
        DepthwiseConv2d_local[1] = (DepthwiseConv2d_local[1] + (PaddedInput_shared_local[1] * compute_shared_local[0]));
      }
    }
  }
  if ((((((int)blockIdx.x) % 17) * 5) + ((((int)threadIdx.x) % 240) / 48)) < 83) {
    compute[(((((((((int)blockIdx.x) / 1632) * 1322688) + ((((int)threadIdx.x) / 240) * 661344)) + (((((int)blockIdx.x) % 1632) / 17) * 6889)) + ((((int)blockIdx.x) % 17) * 415)) + (((((int)threadIdx.x) % 240) / 48) * 83)) + (((int)threadIdx.x) % 48))] = DepthwiseConv2d_local[0];
    if ((((int)threadIdx.x) % 48) < 35) {
      compute[((((((((((int)blockIdx.x) / 1632) * 1322688) + ((((int)threadIdx.x) / 240) * 661344)) + (((((int)blockIdx.x) % 1632) / 17) * 6889)) + ((((int)blockIdx.x) % 17) * 415)) + (((((int)threadIdx.x) % 240) / 48) * 83)) + (((int)threadIdx.x) % 48)) + 48)] = DepthwiseConv2d_local[1];
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int input_size1 = C * KH * KW;
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Bh;
    float *Ad, *Bd, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));
    Bh = (float*)malloc(input_size1 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Bd, input_size1 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        Bh[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, input_size1 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(104448, 1, 1);
    dim3 block(480, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
        hipDeviceSynchronize();
    }
}
