#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float A_shared[4096];
  float A_shared_local[1];
  compute_local[0] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 128; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7))];
    A_shared[(((int)threadIdx.x) + 512)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 65536)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 131072)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 196608)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 262144)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 327680)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 393216)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer * 8)) + (((int)threadIdx.x) & 7)) + 458752)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 8; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 8) + k_inner_outer)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
    }
  }
  compute[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))] = compute_local[0];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 128;
    int block_size = 512;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
