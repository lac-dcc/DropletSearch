#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float A_shared[8192];
  float A_shared_local[4];
  compute_local[0] = 0.000000e+00f;
  compute_local[1] = 0.000000e+00f;
  compute_local[2] = 0.000000e+00f;
  compute_local[3] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31))];
    A_shared[(((int)threadIdx.x) + 64)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    A_shared[(((int)threadIdx.x) + 128)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    A_shared[(((int)threadIdx.x) + 192)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    A_shared[(((int)threadIdx.x) + 256)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    A_shared[(((int)threadIdx.x) + 320)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240)];
    A_shared[(((int)threadIdx.x) + 384)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    A_shared[(((int)threadIdx.x) + 448)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    A_shared[(((int)threadIdx.x) + 512)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
    A_shared[(((int)threadIdx.x) + 576)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432)];
    A_shared[(((int)threadIdx.x) + 640)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
    A_shared[(((int)threadIdx.x) + 704)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 22528)];
    A_shared[(((int)threadIdx.x) + 768)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
    A_shared[(((int)threadIdx.x) + 832)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 26624)];
    A_shared[(((int)threadIdx.x) + 896)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    A_shared[(((int)threadIdx.x) + 960)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 30720)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 32768)];
    A_shared[(((int)threadIdx.x) + 1088)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 34816)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 36864)];
    A_shared[(((int)threadIdx.x) + 1216)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 38912)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 40960)];
    A_shared[(((int)threadIdx.x) + 1344)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 43008)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 45056)];
    A_shared[(((int)threadIdx.x) + 1472)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 47104)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 49152)];
    A_shared[(((int)threadIdx.x) + 1600)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 51200)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 53248)];
    A_shared[(((int)threadIdx.x) + 1728)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 55296)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 57344)];
    A_shared[(((int)threadIdx.x) + 1856)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 59392)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 61440)];
    A_shared[(((int)threadIdx.x) + 1984)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 63488)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 65536)];
    A_shared[(((int)threadIdx.x) + 2112)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 67584)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 69632)];
    A_shared[(((int)threadIdx.x) + 2240)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 71680)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 73728)];
    A_shared[(((int)threadIdx.x) + 2368)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 75776)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 77824)];
    A_shared[(((int)threadIdx.x) + 2496)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 79872)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 81920)];
    A_shared[(((int)threadIdx.x) + 2624)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 83968)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 86016)];
    A_shared[(((int)threadIdx.x) + 2752)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 88064)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 90112)];
    A_shared[(((int)threadIdx.x) + 2880)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 92160)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 94208)];
    A_shared[(((int)threadIdx.x) + 3008)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 96256)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 98304)];
    A_shared[(((int)threadIdx.x) + 3136)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 100352)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 102400)];
    A_shared[(((int)threadIdx.x) + 3264)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 104448)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 106496)];
    A_shared[(((int)threadIdx.x) + 3392)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 108544)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 110592)];
    A_shared[(((int)threadIdx.x) + 3520)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 112640)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 114688)];
    A_shared[(((int)threadIdx.x) + 3648)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 116736)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 118784)];
    A_shared[(((int)threadIdx.x) + 3776)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 120832)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 122880)];
    A_shared[(((int)threadIdx.x) + 3904)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 124928)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 126976)];
    A_shared[(((int)threadIdx.x) + 4032)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 129024)];
    A_shared[(((int)threadIdx.x) + 4096)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 131072)];
    A_shared[(((int)threadIdx.x) + 4160)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 133120)];
    A_shared[(((int)threadIdx.x) + 4224)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 135168)];
    A_shared[(((int)threadIdx.x) + 4288)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 137216)];
    A_shared[(((int)threadIdx.x) + 4352)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 139264)];
    A_shared[(((int)threadIdx.x) + 4416)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 141312)];
    A_shared[(((int)threadIdx.x) + 4480)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 143360)];
    A_shared[(((int)threadIdx.x) + 4544)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 145408)];
    A_shared[(((int)threadIdx.x) + 4608)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 147456)];
    A_shared[(((int)threadIdx.x) + 4672)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 149504)];
    A_shared[(((int)threadIdx.x) + 4736)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 151552)];
    A_shared[(((int)threadIdx.x) + 4800)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 153600)];
    A_shared[(((int)threadIdx.x) + 4864)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 155648)];
    A_shared[(((int)threadIdx.x) + 4928)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 157696)];
    A_shared[(((int)threadIdx.x) + 4992)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 159744)];
    A_shared[(((int)threadIdx.x) + 5056)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 161792)];
    A_shared[(((int)threadIdx.x) + 5120)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 163840)];
    A_shared[(((int)threadIdx.x) + 5184)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 165888)];
    A_shared[(((int)threadIdx.x) + 5248)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 167936)];
    A_shared[(((int)threadIdx.x) + 5312)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 169984)];
    A_shared[(((int)threadIdx.x) + 5376)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 172032)];
    A_shared[(((int)threadIdx.x) + 5440)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 174080)];
    A_shared[(((int)threadIdx.x) + 5504)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 176128)];
    A_shared[(((int)threadIdx.x) + 5568)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 178176)];
    A_shared[(((int)threadIdx.x) + 5632)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 180224)];
    A_shared[(((int)threadIdx.x) + 5696)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 182272)];
    A_shared[(((int)threadIdx.x) + 5760)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 184320)];
    A_shared[(((int)threadIdx.x) + 5824)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 186368)];
    A_shared[(((int)threadIdx.x) + 5888)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 188416)];
    A_shared[(((int)threadIdx.x) + 5952)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 190464)];
    A_shared[(((int)threadIdx.x) + 6016)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 192512)];
    A_shared[(((int)threadIdx.x) + 6080)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 194560)];
    A_shared[(((int)threadIdx.x) + 6144)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 196608)];
    A_shared[(((int)threadIdx.x) + 6208)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 198656)];
    A_shared[(((int)threadIdx.x) + 6272)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 200704)];
    A_shared[(((int)threadIdx.x) + 6336)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 202752)];
    A_shared[(((int)threadIdx.x) + 6400)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 204800)];
    A_shared[(((int)threadIdx.x) + 6464)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 206848)];
    A_shared[(((int)threadIdx.x) + 6528)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 208896)];
    A_shared[(((int)threadIdx.x) + 6592)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 210944)];
    A_shared[(((int)threadIdx.x) + 6656)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 212992)];
    A_shared[(((int)threadIdx.x) + 6720)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 215040)];
    A_shared[(((int)threadIdx.x) + 6784)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 217088)];
    A_shared[(((int)threadIdx.x) + 6848)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 219136)];
    A_shared[(((int)threadIdx.x) + 6912)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 221184)];
    A_shared[(((int)threadIdx.x) + 6976)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 223232)];
    A_shared[(((int)threadIdx.x) + 7040)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 225280)];
    A_shared[(((int)threadIdx.x) + 7104)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 227328)];
    A_shared[(((int)threadIdx.x) + 7168)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 229376)];
    A_shared[(((int)threadIdx.x) + 7232)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 231424)];
    A_shared[(((int)threadIdx.x) + 7296)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 233472)];
    A_shared[(((int)threadIdx.x) + 7360)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 235520)];
    A_shared[(((int)threadIdx.x) + 7424)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 237568)];
    A_shared[(((int)threadIdx.x) + 7488)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 239616)];
    A_shared[(((int)threadIdx.x) + 7552)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 241664)];
    A_shared[(((int)threadIdx.x) + 7616)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 243712)];
    A_shared[(((int)threadIdx.x) + 7680)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 245760)];
    A_shared[(((int)threadIdx.x) + 7744)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 247808)];
    A_shared[(((int)threadIdx.x) + 7808)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 249856)];
    A_shared[(((int)threadIdx.x) + 7872)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 251904)];
    A_shared[(((int)threadIdx.x) + 7936)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 253952)];
    A_shared[(((int)threadIdx.x) + 8000)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 256000)];
    A_shared[(((int)threadIdx.x) + 8064)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 258048)];
    A_shared[(((int)threadIdx.x) + 8128)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 260096)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 32; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 32) + k_inner_outer)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.x) * 32) + k_inner_outer) + 2048)];
      A_shared_local[2] = A_shared[(((((int)threadIdx.x) * 32) + k_inner_outer) + 4096)];
      A_shared_local[3] = A_shared[(((((int)threadIdx.x) * 32) + k_inner_outer) + 6144)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
      compute_local[1] = (compute_local[1] + A_shared_local[1]);
      compute_local[2] = (compute_local[2] + A_shared_local[2]);
      compute_local[3] = (compute_local[3] + A_shared_local[3]);
    }
  }
  compute[((((int)blockIdx.x) * 256) + ((int)threadIdx.x))] = compute_local[0];
  compute[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) + 64)] = compute_local[1];
  compute[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) + 128)] = compute_local[2];
  compute[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) + 192)] = compute_local[3];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 256;
    int block_size = 64;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
