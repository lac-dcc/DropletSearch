#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 96;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 165, KH = 7;
int NW = 165, KW = 7;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(288) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ kernel) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[2907];
  __shared__ float compute_shared[49];
  float PaddedInput_shared_local[2];
  float compute_shared_local[1];
  DepthwiseConv2d_local[0] = 0.000000e+00f;
  DepthwiseConv2d_local[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = ((((1 <= (((int)blockIdx.x) % 14)) && (3 <= (((int)threadIdx.x) % 171))) && ((((int)threadIdx.x) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + ((((int)threadIdx.x) / 171) * 165)) + (((int)threadIdx.x) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 288)] = ((((1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 288) / 513))) && (1 <= (((((int)threadIdx.x) / 3) + 39) % 57))) && (((((int)threadIdx.x) + 117) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 288) / 171) * 165)) + ((((int)threadIdx.x) + 117) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 576)] = (((1 <= (((((int)threadIdx.x) / 3) + 21) % 57)) && (((((int)threadIdx.x) + 63) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 576) / 171) * 165)) + ((((int)threadIdx.x) + 63) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 864)] = (((1 <= (((((int)threadIdx.x) / 3) + 3) % 57)) && (((((int)threadIdx.x) + 9) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 864) / 171) * 165)) + ((((int)threadIdx.x) + 9) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1152)] = (((1 <= (((((int)threadIdx.x) / 3) + 42) % 57)) && (((((int)threadIdx.x) + 126) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 1152) / 171) * 165)) + ((((int)threadIdx.x) + 126) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1440)] = (((1 <= (((((int)threadIdx.x) / 3) + 24) % 57)) && (((((int)threadIdx.x) + 72) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 1440) / 171) * 165)) + ((((int)threadIdx.x) + 72) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1728)] = (((1 <= (((((int)threadIdx.x) / 3) + 6) % 57)) && (((((int)threadIdx.x) + 18) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 1728) / 171) * 165)) + ((((int)threadIdx.x) + 18) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2016)] = (((((((((int)threadIdx.x) + 2016) / 2052) + (((int)blockIdx.x) % 14)) < 14) && (1 <= (((((int)threadIdx.x) / 3) + 45) % 57))) && (((((int)threadIdx.x) + 135) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 2016) / 171) * 165)) + ((((int)threadIdx.x) + 135) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2304) / 513)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 2304)] = (((((((((int)threadIdx.x) + 2304) / 2052) + (((int)blockIdx.x) % 14)) < 14) && (1 <= (((((int)threadIdx.x) / 3) + 27) % 57))) && (((((int)threadIdx.x) + 81) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 2304) / 171) * 165)) + ((((int)threadIdx.x) + 81) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2592) / 513)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 2592)] = (((((((((int)threadIdx.x) + 2592) / 2052) + (((int)blockIdx.x) % 14)) < 14) && (1 <= (((((int)threadIdx.x) / 3) + 9) % 57))) && (((((int)threadIdx.x) + 27) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 2592) / 171) * 165)) + ((((int)threadIdx.x) + 27) % 171)) - 498)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 27) {
    if ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2880) / 513)) < 57) {
      PaddedInput_shared[(((int)threadIdx.x) + 2880)] = ((((((((int)threadIdx.x) + 2880) / 2052) + (((int)blockIdx.x) % 14)) < 14) && (((int)threadIdx.x) < 24)) ? data[((((((((int)blockIdx.x) / 14) * 27225) + ((((int)blockIdx.x) % 14) * 1980)) + (((((int)threadIdx.x) + 2880) / 171) * 165)) + ((int)threadIdx.x)) - 354)] : 0.000000e+00f);
    }
  }
  if (((int)threadIdx.x) < 49) {
    compute_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) % 1344) / 14) * 49) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner_outer = 0; k_inner_outer < 49; ++k_inner_outer) {
    if ((((((int)blockIdx.x) % 14) * 4) + ((((((int)threadIdx.x) / 48) * 2) + (k_inner_outer / 7)) / 3)) < 57) {
      PaddedInput_shared_local[0] = PaddedInput_shared[(((((((int)threadIdx.x) / 48) * 342) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7))];
      if ((((((int)threadIdx.x) % 48) * 2) + (k_inner_outer % 7)) < 75) {
        PaddedInput_shared_local[1] = PaddedInput_shared[((((((((int)threadIdx.x) / 48) * 342) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7)) + 96)];
      }
    }
    compute_shared_local[0] = compute_shared[k_inner_outer];
    if ((((((int)blockIdx.x) % 14) * 6) + (((int)threadIdx.x) / 48)) < 83) {
      DepthwiseConv2d_local[0] = (DepthwiseConv2d_local[0] + (PaddedInput_shared_local[0] * compute_shared_local[0]));
      if ((((int)threadIdx.x) % 48) < 35) {
        DepthwiseConv2d_local[1] = (DepthwiseConv2d_local[1] + (PaddedInput_shared_local[1] * compute_shared_local[0]));
      }
    }
  }
  if ((((((int)blockIdx.x) % 14) * 6) + (((int)threadIdx.x) / 48)) < 83) {
    compute[(((((((int)blockIdx.x) / 14) * 6889) + ((((int)blockIdx.x) % 14) * 498)) + ((((int)threadIdx.x) / 48) * 83)) + (((int)threadIdx.x) % 48))] = DepthwiseConv2d_local[0];
    if ((((int)threadIdx.x) % 48) < 35) {
      compute[((((((((int)blockIdx.x) / 14) * 6889) + ((((int)blockIdx.x) % 14) * 498)) + ((((int)threadIdx.x) / 48) * 83)) + (((int)threadIdx.x) % 48)) + 48)] = DepthwiseConv2d_local[1];
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int input_size1 = C * KH * KW;
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Bh;
    float *Ad, *Bd, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));
    Bh = (float*)malloc(input_size1 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Bd, input_size1 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        Bh[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, input_size1 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(172032, 1, 1);
    dim3 block(288, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
        hipDeviceSynchronize();
    }
}
