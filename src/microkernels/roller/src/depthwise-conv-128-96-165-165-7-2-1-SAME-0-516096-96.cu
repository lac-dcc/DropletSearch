#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 96;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 165, KH = 7;
int NW = 165, KW = 7;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(96) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ kernel) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[1539];
  __shared__ float compute_shared[49];
  float PaddedInput_shared_local[2];
  float compute_shared_local[1];
  DepthwiseConv2d_local[0] = 0.000000e+00f;
  DepthwiseConv2d_local[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((0 < (((int)blockIdx.x) % 42)) && (3 <= ((int)threadIdx.x))) ? data[(((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + ((int)threadIdx.x)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 96)] = ((((3 <= (((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 96) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 32) % 57))) && (((((int)threadIdx.x) + 96) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 96) / 171) * 165)) + ((((int)threadIdx.x) + 96) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 192)] = ((3 <= (((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 192) / 171))) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 192) / 171) * 165)) + ((int)threadIdx.x)) - 477)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 288)] = ((((3 <= (((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 288) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 39) % 57))) && (((((int)threadIdx.x) + 117) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 288) / 171) * 165)) + ((((int)threadIdx.x) + 117) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 384)] = ((3 <= (((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 384) / 171))) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 384) / 171) * 165)) + ((int)threadIdx.x)) - 456)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 480)] = ((((3 <= (((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 480) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 46) % 57))) && (((((int)threadIdx.x) + 138) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 480) / 171) * 165)) + ((((int)threadIdx.x) + 138) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 576)] = data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 576) / 171) * 165)) + ((int)threadIdx.x)) - 435)];
  PaddedInput_shared[(((int)threadIdx.x) + 672)] = (((((((((int)threadIdx.x) + 672) / 684) + (((int)blockIdx.x) % 42)) < 42) && (1 <= (((((int)threadIdx.x) / 3) + 53) % 57))) && (((((int)threadIdx.x) + 159) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 672) / 171) * 165)) + ((((int)threadIdx.x) + 159) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 768)] = (((((((((int)threadIdx.x) + 768) / 684) + (((int)blockIdx.x) % 42)) < 42) && (1 <= (((((int)threadIdx.x) / 3) + 28) % 57))) && (((((int)threadIdx.x) + 84) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 768) / 171) * 165)) + ((((int)threadIdx.x) + 84) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 864)] = (((((((int)threadIdx.x) + 864) / 684) + (((int)blockIdx.x) % 42)) < 42) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 864) / 171) * 165)) + ((int)threadIdx.x)) - 489)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 960)] = (((((((((int)threadIdx.x) + 960) / 684) + (((int)blockIdx.x) % 42)) < 42) && (1 <= (((((int)threadIdx.x) / 3) + 35) % 57))) && (((((int)threadIdx.x) + 105) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 960) / 171) * 165)) + ((((int)threadIdx.x) + 105) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1056)] = (((((((int)threadIdx.x) + 1056) / 684) + (((int)blockIdx.x) % 42)) < 42) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 1056) / 171) * 165)) + ((int)threadIdx.x)) - 468)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 1152) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1152)] = (((((((((int)threadIdx.x) + 1152) / 684) + (((int)blockIdx.x) % 42)) < 42) && (1 <= (((((int)threadIdx.x) / 3) + 42) % 57))) && (((((int)threadIdx.x) + 126) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 1152) / 171) * 165)) + ((((int)threadIdx.x) + 126) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 1248) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1248)] = (((((((int)threadIdx.x) + 1248) / 684) + (((int)blockIdx.x) % 42)) < 42) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 1248) / 171) * 165)) + ((int)threadIdx.x)) - 447)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 1344) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((((((((int)threadIdx.x) + 1344) / 684) + (((int)blockIdx.x) % 42)) < 42) && (1 <= (((((int)threadIdx.x) / 3) + 49) % 57))) && (((((int)threadIdx.x) + 147) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 1344) / 171) * 165)) + ((((int)threadIdx.x) + 147) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) + 1440) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1440)] = (((((((int)threadIdx.x) + 1440) / 684) + (((int)blockIdx.x) % 42)) < 42) ? data[((((((((int)blockIdx.x) / 42) * 27225) + ((((int)blockIdx.x) % 42) * 660)) + (((((int)threadIdx.x) + 1440) / 171) * 165)) + ((int)threadIdx.x)) - 426)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 3) {
    if ((((int)blockIdx.x) % 42) < 41) {
      PaddedInput_shared[(((int)threadIdx.x) + 1536)] = 0.000000e+00f;
    }
  }
  if (((int)threadIdx.x) < 49) {
    compute_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) % 4032) / 42) * 49) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner_outer = 0; k_inner_outer < 49; ++k_inner_outer) {
    if (((((((int)blockIdx.x) % 42) * 4) + ((((int)threadIdx.x) / 48) * 2)) + (k_inner_outer / 7)) < 171) {
      PaddedInput_shared_local[0] = PaddedInput_shared[(((((((int)threadIdx.x) / 48) * 342) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7))];
      if ((((((int)threadIdx.x) % 48) * 2) + (k_inner_outer % 7)) < 75) {
        PaddedInput_shared_local[1] = PaddedInput_shared[((((((((int)threadIdx.x) / 48) * 342) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7)) + 96)];
      }
    }
    compute_shared_local[0] = compute_shared[k_inner_outer];
    if ((((((int)blockIdx.x) % 42) * 2) + (((int)threadIdx.x) / 48)) < 83) {
      DepthwiseConv2d_local[0] = (DepthwiseConv2d_local[0] + (PaddedInput_shared_local[0] * compute_shared_local[0]));
      if ((((int)threadIdx.x) % 48) < 35) {
        DepthwiseConv2d_local[1] = (DepthwiseConv2d_local[1] + (PaddedInput_shared_local[1] * compute_shared_local[0]));
      }
    }
  }
  if ((((((int)blockIdx.x) % 42) * 2) + (((int)threadIdx.x) / 48)) < 83) {
    compute[(((((((int)blockIdx.x) / 42) * 6889) + ((((int)blockIdx.x) % 42) * 166)) + ((((int)threadIdx.x) / 48) * 83)) + (((int)threadIdx.x) % 48))] = DepthwiseConv2d_local[0];
    if ((((int)threadIdx.x) % 48) < 35) {
      compute[((((((((int)blockIdx.x) / 42) * 6889) + ((((int)blockIdx.x) % 42) * 166)) + ((((int)threadIdx.x) / 48) * 83)) + (((int)threadIdx.x) % 48)) + 48)] = DepthwiseConv2d_local[1];
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int input_size1 = C * KH * KW;
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Bh;
    float *Ad, *Bd, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));
    Bh = (float*)malloc(input_size1 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Bd, input_size1 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        Bh[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, input_size1 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(516096, 1, 1);
    dim3 block(96, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
        hipDeviceSynchronize();
    }
}
