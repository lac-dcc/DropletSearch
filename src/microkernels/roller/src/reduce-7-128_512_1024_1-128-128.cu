#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float A_shared[8192];
  float A_shared_local[4];
  compute_local[0] = 0.000000e+00f;
  compute_local[1] = 0.000000e+00f;
  compute_local[2] = 0.000000e+00f;
  compute_local[3] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15))];
    A_shared[(((int)threadIdx.x) + 128)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 8192)];
    A_shared[(((int)threadIdx.x) + 256)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 16384)];
    A_shared[(((int)threadIdx.x) + 384)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 24576)];
    A_shared[(((int)threadIdx.x) + 512)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 32768)];
    A_shared[(((int)threadIdx.x) + 640)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 40960)];
    A_shared[(((int)threadIdx.x) + 768)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 49152)];
    A_shared[(((int)threadIdx.x) + 896)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 57344)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 65536)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 73728)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 81920)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 90112)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 98304)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 106496)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 114688)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 122880)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 131072)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 139264)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 147456)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 155648)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 163840)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 172032)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 180224)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 188416)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 196608)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 204800)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 212992)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 221184)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 229376)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 237568)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 245760)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 253952)];
    A_shared[(((int)threadIdx.x) + 4096)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 262144)];
    A_shared[(((int)threadIdx.x) + 4224)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 270336)];
    A_shared[(((int)threadIdx.x) + 4352)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 278528)];
    A_shared[(((int)threadIdx.x) + 4480)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 286720)];
    A_shared[(((int)threadIdx.x) + 4608)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 294912)];
    A_shared[(((int)threadIdx.x) + 4736)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 303104)];
    A_shared[(((int)threadIdx.x) + 4864)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 311296)];
    A_shared[(((int)threadIdx.x) + 4992)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 319488)];
    A_shared[(((int)threadIdx.x) + 5120)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 327680)];
    A_shared[(((int)threadIdx.x) + 5248)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 335872)];
    A_shared[(((int)threadIdx.x) + 5376)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 344064)];
    A_shared[(((int)threadIdx.x) + 5504)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 352256)];
    A_shared[(((int)threadIdx.x) + 5632)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 360448)];
    A_shared[(((int)threadIdx.x) + 5760)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 368640)];
    A_shared[(((int)threadIdx.x) + 5888)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 376832)];
    A_shared[(((int)threadIdx.x) + 6016)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 385024)];
    A_shared[(((int)threadIdx.x) + 6144)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 393216)];
    A_shared[(((int)threadIdx.x) + 6272)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 401408)];
    A_shared[(((int)threadIdx.x) + 6400)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 409600)];
    A_shared[(((int)threadIdx.x) + 6528)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 417792)];
    A_shared[(((int)threadIdx.x) + 6656)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 425984)];
    A_shared[(((int)threadIdx.x) + 6784)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 434176)];
    A_shared[(((int)threadIdx.x) + 6912)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 442368)];
    A_shared[(((int)threadIdx.x) + 7040)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 450560)];
    A_shared[(((int)threadIdx.x) + 7168)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 458752)];
    A_shared[(((int)threadIdx.x) + 7296)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 466944)];
    A_shared[(((int)threadIdx.x) + 7424)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 475136)];
    A_shared[(((int)threadIdx.x) + 7552)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 483328)];
    A_shared[(((int)threadIdx.x) + 7680)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 491520)];
    A_shared[(((int)threadIdx.x) + 7808)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 499712)];
    A_shared[(((int)threadIdx.x) + 7936)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 507904)];
    A_shared[(((int)threadIdx.x) + 8064)] = A[(((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer * 16)) + (((int)threadIdx.x) & 15)) + 516096)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 16; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 16) + k_inner_outer)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.x) * 16) + k_inner_outer) + 2048)];
      A_shared_local[2] = A_shared[(((((int)threadIdx.x) * 16) + k_inner_outer) + 4096)];
      A_shared_local[3] = A_shared[(((((int)threadIdx.x) * 16) + k_inner_outer) + 6144)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
      compute_local[1] = (compute_local[1] + A_shared_local[1]);
      compute_local[2] = (compute_local[2] + A_shared_local[2]);
      compute_local[3] = (compute_local[3] + A_shared_local[3]);
    }
  }
  compute[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))] = compute_local[0];
  compute[(((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) + 128)] = compute_local[1];
  compute[(((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) + 256)] = compute_local[2];
  compute[(((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) + 384)] = compute_local[3];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 128;
    int block_size = 128;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
