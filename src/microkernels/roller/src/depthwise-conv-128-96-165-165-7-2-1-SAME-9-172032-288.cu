#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 96;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 165, KH = 7;
int NW = 165, KW = 7;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(288) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ kernel) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[3762];
  __shared__ float compute_shared[98];
  float PaddedInput_shared_local[2];
  float compute_shared_local[1];
  DepthwiseConv2d_local[0] = 0.000000e+00f;
  DepthwiseConv2d_local[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = ((((1 <= (((int)blockIdx.x) % 28)) && (3 <= (((int)threadIdx.x) % 171))) && ((((int)threadIdx.x) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + ((((int)threadIdx.x) / 171) * 165)) + (((int)threadIdx.x) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 288)] = ((((1 <= (((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 288) / 513))) && (1 <= (((((int)threadIdx.x) / 3) + 39) % 57))) && (((((int)threadIdx.x) + 117) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 288) / 171) * 165)) + ((((int)threadIdx.x) + 117) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 576)] = (((1 <= (((((int)threadIdx.x) / 3) + 21) % 57)) && (((((int)threadIdx.x) + 63) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 576) / 171) * 165)) + ((((int)threadIdx.x) + 63) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 864)] = (((((((((int)threadIdx.x) + 864) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 3) % 57))) && (((((int)threadIdx.x) + 9) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 864) / 171) * 165)) + ((((int)threadIdx.x) + 9) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1152)] = (((((((((int)threadIdx.x) + 1152) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 42) % 57))) && (((((int)threadIdx.x) + 126) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 1152) / 171) * 165)) + ((((int)threadIdx.x) + 126) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 1440) / 513)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 1440)] = (((((((((int)threadIdx.x) + 1440) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 24) % 57))) && (((((int)threadIdx.x) + 72) % 171) < 168)) ? data[((((((((int)blockIdx.x) / 28) * 54450) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 1440) / 171) * 165)) + ((((int)threadIdx.x) + 72) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 28) * 2) + ((((((int)threadIdx.x) / 9) + 192) % 209) / 57)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 1728)] = (((((1 <= (((((int)blockIdx.x) % 28) * 2) + ((((((int)threadIdx.x) / 9) + 192) % 209) / 57))) && ((((((((int)threadIdx.x) / 9) + 192) % 209) / 114) + (((int)blockIdx.x) % 28)) < 28)) && (1 <= (((((int)threadIdx.x) / 3) + 6) % 57))) && (((((int)threadIdx.x) + 18) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 1728) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((((int)threadIdx.x) / 9) + 192) % 209) / 19) * 165)) + ((((int)threadIdx.x) + 18) % 171)) - 498)] : 0.000000e+00f);
  }
  PaddedInput_shared[(((int)threadIdx.x) + 2016)] = ((((0 < (((int)blockIdx.x) % 28)) && (1 <= (((((int)threadIdx.x) / 3) + 45) % 57))) && (((((int)threadIdx.x) + 135) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 2016) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 135) / 171) * 165)) + ((((int)threadIdx.x) + 135) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2304)] = ((((1 <= (((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 423) / 513))) && (1 <= (((((int)threadIdx.x) / 3) + 27) % 57))) && (((((int)threadIdx.x) + 81) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 2304) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 423) / 171) * 165)) + ((((int)threadIdx.x) + 81) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2592)] = (((1 <= (((((int)threadIdx.x) / 3) + 9) % 57)) && (((((int)threadIdx.x) + 27) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 2592) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 711) / 171) * 165)) + ((((int)threadIdx.x) + 27) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2880)] = (((((((((int)threadIdx.x) + 999) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 48) % 57))) && (((((int)threadIdx.x) + 144) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 2880) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 999) / 171) * 165)) + ((((int)threadIdx.x) + 144) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 1287) / 513)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 3168)] = (((((((((int)threadIdx.x) + 1287) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 30) % 57))) && (((((int)threadIdx.x) + 90) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 3168) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 1287) / 171) * 165)) + ((((int)threadIdx.x) + 90) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 1575) / 513)) < 57) {
    PaddedInput_shared[(((int)threadIdx.x) + 3456)] = (((((((((int)threadIdx.x) + 1575) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (1 <= (((((int)threadIdx.x) / 3) + 12) % 57))) && (((((int)threadIdx.x) + 36) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 3456) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 1575) / 171) * 165)) + ((((int)threadIdx.x) + 36) % 171)) - 498)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 18) {
    if ((((((int)blockIdx.x) % 28) * 2) + ((((int)threadIdx.x) + 1863) / 513)) < 57) {
      PaddedInput_shared[(((int)threadIdx.x) + 3744)] = ((((((((int)threadIdx.x) + 1863) / 1026) + (((int)blockIdx.x) % 28)) < 28) && (((int)threadIdx.x) < 15)) ? data[(((((((((int)blockIdx.x) / 28) * 54450) + (((((int)threadIdx.x) + 3744) / 1881) * 27225)) + ((((int)blockIdx.x) % 28) * 990)) + (((((int)threadIdx.x) + 1863) / 171) * 165)) + ((int)threadIdx.x)) - 345)] : 0.000000e+00f);
    }
  }
  if (((int)threadIdx.x) < 98) {
    compute_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) % 1344) / 28) * 98) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner_outer = 0; k_inner_outer < 49; ++k_inner_outer) {
    if ((((((int)blockIdx.x) % 28) * 2) + (((((((int)threadIdx.x) % 144) / 48) * 2) + (k_inner_outer / 7)) / 3)) < 57) {
      PaddedInput_shared_local[0] = PaddedInput_shared[((((((((int)threadIdx.x) / 144) * 1881) + (((((int)threadIdx.x) % 144) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7))];
      if ((((((int)threadIdx.x) % 48) * 2) + (k_inner_outer % 7)) < 75) {
        PaddedInput_shared_local[1] = PaddedInput_shared[(((((((((int)threadIdx.x) / 144) * 1881) + (((((int)threadIdx.x) % 144) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7)) + 96)];
      }
    }
    compute_shared_local[0] = compute_shared[(((((int)threadIdx.x) / 144) * 49) + k_inner_outer)];
    if ((((((int)blockIdx.x) % 28) * 3) + ((((int)threadIdx.x) % 144) / 48)) < 83) {
      DepthwiseConv2d_local[0] = (DepthwiseConv2d_local[0] + (PaddedInput_shared_local[0] * compute_shared_local[0]));
      if ((((int)threadIdx.x) % 48) < 35) {
        DepthwiseConv2d_local[1] = (DepthwiseConv2d_local[1] + (PaddedInput_shared_local[1] * compute_shared_local[0]));
      }
    }
  }
  if ((((((int)blockIdx.x) % 28) * 3) + ((((int)threadIdx.x) % 144) / 48)) < 83) {
    compute[((((((((int)blockIdx.x) / 28) * 13778) + ((((int)threadIdx.x) / 144) * 6889)) + ((((int)blockIdx.x) % 28) * 249)) + (((((int)threadIdx.x) % 144) / 48) * 83)) + (((int)threadIdx.x) % 48))] = DepthwiseConv2d_local[0];
    if ((((int)threadIdx.x) % 48) < 35) {
      compute[(((((((((int)blockIdx.x) / 28) * 13778) + ((((int)threadIdx.x) / 144) * 6889)) + ((((int)blockIdx.x) % 28) * 249)) + (((((int)threadIdx.x) % 144) / 48) * 83)) + (((int)threadIdx.x) % 48)) + 48)] = DepthwiseConv2d_local[1];
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int input_size1 = C * KH * KW;
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Bh;
    float *Ad, *Bd, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));
    Bh = (float*)malloc(input_size1 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Bd, input_size1 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        Bh[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, input_size1 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(172032, 1, 1);
    dim3 block(288, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
        hipDeviceSynchronize();
    }
}
