#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 96;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 165, KH = 7;
int NW = 165, KW = 7;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(384) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ kernel) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[4446];
  __shared__ float compute_shared[49];
  float PaddedInput_shared_local[2];
  float compute_shared_local[1];
  DepthwiseConv2d_local[0] = 0.000000e+00f;
  DepthwiseConv2d_local[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = ((((3 <= (((((int)blockIdx.x) % 21) * 8) + (((int)threadIdx.x) / 171))) && (3 <= (((int)threadIdx.x) % 171))) && ((((int)threadIdx.x) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + ((((int)threadIdx.x) / 171) * 165)) + (((int)threadIdx.x) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 384)] = ((((3 <= (((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 384) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 14) % 57))) && (((((int)threadIdx.x) + 42) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 384) / 171) * 165)) + ((((int)threadIdx.x) + 42) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 768)] = (((1 <= (((((int)threadIdx.x) / 3) + 28) % 57)) && (((((int)threadIdx.x) + 84) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 768) / 171) * 165)) + ((((int)threadIdx.x) + 84) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1152)] = (((((((((int)threadIdx.x) + 1152) / 1368) + (((int)blockIdx.x) % 21)) < 21) && (1 <= (((((int)threadIdx.x) / 3) + 42) % 57))) && (((((int)threadIdx.x) + 126) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 1152) / 171) * 165)) + ((((int)threadIdx.x) + 126) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 1536) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1536)] = (((((((((int)threadIdx.x) + 1536) / 1368) + (((int)blockIdx.x) % 21)) < 21) && (1 <= (((((int)threadIdx.x) / 3) + 56) % 57))) && (((((int)threadIdx.x) + 168) % 171) < 168)) ? data[(((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 1536) / 171) * 165)) + ((((int)threadIdx.x) + 168) % 171)) - 498)] : 0.000000e+00f);
  }
  if ((((((int)blockIdx.x) % 21) * 8) + ((((((int)threadIdx.x) / 3) + 640) % 741) / 57)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 1920)] = (((((3 <= (((((int)blockIdx.x) % 21) * 8) + ((((((int)threadIdx.x) / 3) + 640) % 741) / 57))) && ((((((((int)threadIdx.x) / 3) + 640) % 741) / 456) + (((int)blockIdx.x) % 21)) < 21)) && (1 <= (((((int)threadIdx.x) / 3) + 13) % 57))) && (((((int)threadIdx.x) + 39) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 1920) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((((int)threadIdx.x) / 3) + 640) % 741) / 57) * 165)) + ((((int)threadIdx.x) + 39) % 171)) - 498)] : 0.000000e+00f);
  }
  PaddedInput_shared[(((int)threadIdx.x) + 2304)] = ((((3 <= (((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 81) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 27) % 57))) && (((((int)threadIdx.x) + 81) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 2304) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 81) / 171) * 165)) + ((((int)threadIdx.x) + 81) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((((3 <= (((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 465) / 171))) && (1 <= (((((int)threadIdx.x) / 3) + 41) % 57))) && (((((int)threadIdx.x) + 123) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 2688) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 465) / 171) * 165)) + ((((int)threadIdx.x) + 123) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3072)] = (((1 <= (((((int)threadIdx.x) / 3) + 55) % 57)) && (((((int)threadIdx.x) + 165) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 3072) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 849) / 171) * 165)) + ((((int)threadIdx.x) + 165) % 171)) - 498)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3456)] = (((((((((int)threadIdx.x) + 1233) / 1368) + (((int)blockIdx.x) % 21)) < 21) && (1 <= (((((int)threadIdx.x) / 3) + 12) % 57))) && (((((int)threadIdx.x) + 36) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 3456) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 1233) / 171) * 165)) + ((((int)threadIdx.x) + 36) % 171)) - 498)] : 0.000000e+00f);
  if ((((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 1617) / 171)) < 171) {
    PaddedInput_shared[(((int)threadIdx.x) + 3840)] = (((((((((int)threadIdx.x) + 1617) / 1368) + (((int)blockIdx.x) % 21)) < 21) && (1 <= (((((int)threadIdx.x) / 3) + 26) % 57))) && (((((int)threadIdx.x) + 78) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 3840) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 1617) / 171) * 165)) + ((((int)threadIdx.x) + 78) % 171)) - 498)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 222) {
    if ((((((int)blockIdx.x) % 21) * 8) + ((((int)threadIdx.x) + 2001) / 171)) < 171) {
      PaddedInput_shared[(((int)threadIdx.x) + 4224)] = (((((((((int)threadIdx.x) + 2001) / 1368) + (((int)blockIdx.x) % 21)) < 21) && (1 <= (((((int)threadIdx.x) / 3) + 40) % 57))) && (((((int)threadIdx.x) + 120) % 171) < 168)) ? data[((((((((((int)blockIdx.x) / 2016) * 5227200) + (((((int)threadIdx.x) + 4224) / 2223) * 2613600)) + (((((int)blockIdx.x) % 2016) / 21) * 27225)) + ((((int)blockIdx.x) % 21) * 1320)) + (((((int)threadIdx.x) + 2001) / 171) * 165)) + ((((int)threadIdx.x) + 120) % 171)) - 498)] : 0.000000e+00f);
    }
  }
  if (((int)threadIdx.x) < 49) {
    compute_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) % 2016) / 21) * 49) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner_outer = 0; k_inner_outer < 49; ++k_inner_outer) {
    if (((((((int)blockIdx.x) % 21) * 8) + (((((int)threadIdx.x) % 192) / 48) * 2)) + (k_inner_outer / 7)) < 171) {
      PaddedInput_shared_local[0] = PaddedInput_shared[((((((((int)threadIdx.x) / 192) * 2223) + (((((int)threadIdx.x) % 192) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7))];
      if ((((((int)threadIdx.x) % 48) * 2) + (k_inner_outer % 7)) < 75) {
        PaddedInput_shared_local[1] = PaddedInput_shared[(((((((((int)threadIdx.x) / 192) * 2223) + (((((int)threadIdx.x) % 192) / 48) * 342)) + ((k_inner_outer / 7) * 171)) + ((((int)threadIdx.x) % 48) * 2)) + (k_inner_outer % 7)) + 96)];
      }
    }
    compute_shared_local[0] = compute_shared[k_inner_outer];
    if ((((((int)blockIdx.x) % 21) * 4) + ((((int)threadIdx.x) % 192) / 48)) < 83) {
      DepthwiseConv2d_local[0] = (DepthwiseConv2d_local[0] + (PaddedInput_shared_local[0] * compute_shared_local[0]));
      if ((((int)threadIdx.x) % 48) < 35) {
        DepthwiseConv2d_local[1] = (DepthwiseConv2d_local[1] + (PaddedInput_shared_local[1] * compute_shared_local[0]));
      }
    }
  }
  if ((((((int)blockIdx.x) % 21) * 4) + ((((int)threadIdx.x) % 192) / 48)) < 83) {
    compute[(((((((((int)blockIdx.x) / 2016) * 1322688) + ((((int)threadIdx.x) / 192) * 661344)) + (((((int)blockIdx.x) % 2016) / 21) * 6889)) + ((((int)blockIdx.x) % 21) * 332)) + (((((int)threadIdx.x) % 192) / 48) * 83)) + (((int)threadIdx.x) % 48))] = DepthwiseConv2d_local[0];
    if ((((int)threadIdx.x) % 48) < 35) {
      compute[((((((((((int)blockIdx.x) / 2016) * 1322688) + ((((int)threadIdx.x) / 192) * 661344)) + (((((int)blockIdx.x) % 2016) / 21) * 6889)) + ((((int)blockIdx.x) % 21) * 332)) + (((((int)threadIdx.x) % 192) / 48) * 83)) + (((int)threadIdx.x) % 48)) + 48)] = DepthwiseConv2d_local[1];
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int input_size1 = C * KH * KW;
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Bh;
    float *Ad, *Bd, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));
    Bh = (float*)malloc(input_size1 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Bd, input_size1 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        Bh[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, input_size1 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(129024, 1, 1);
    dim3 block(384, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
        hipDeviceSynchronize();
    }
}
