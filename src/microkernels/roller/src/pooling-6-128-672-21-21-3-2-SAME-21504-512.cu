#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int N = 128;
int C = 672;
std::string P = "SAME";
int S_height = 2, S_width = 2;
int NH = 21, KH = 3;
int NW = 21, KW = 3;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) default_function_kernel(float* __restrict__ Pool2d, float* __restrict__ data) {
  if ((((int)threadIdx.x) & 127) < 88) {
    Pool2d[((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7))] = 0.000000e+00f;
    if ((((int)threadIdx.x) & 7) < 3) {
      Pool2d[(((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7)) + 8)] = 0.000000e+00f;
    }
  }
  for (int k_inner_outer = 0; k_inner_outer < 9; ++k_inner_outer) {
    if ((((int)threadIdx.x) & 127) < 88) {
      Pool2d[((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7))] = (Pool2d[((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7))] + (data[((((((((int)blockIdx.x) * 2116) + ((((int)threadIdx.x) >> 7) * 529)) + (((((int)threadIdx.x) & 127) >> 3) * 46)) + ((k_inner_outer / 3) * 23)) + ((((int)threadIdx.x) & 7) * 2)) + (k_inner_outer % 3))] * 1.111111e-01f));
      if ((((int)threadIdx.x) & 7) < 3) {
        Pool2d[(((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7)) + 8)] = (Pool2d[(((((((int)blockIdx.x) * 484) + ((((int)threadIdx.x) >> 7) * 121)) + (((((int)threadIdx.x) & 127) >> 3) * 11)) + (((int)threadIdx.x) & 7)) + 8)] + (data[(((((((((int)blockIdx.x) * 2116) + ((((int)threadIdx.x) >> 7) * 529)) + (((((int)threadIdx.x) & 127) >> 3) * 46)) + ((k_inner_outer / 3) * 23)) + ((((int)threadIdx.x) & 7) * 2)) + (k_inner_outer % 3)) + 16)] * 1.111111e-01f));
      }
    }
  }
}

int main(int argc, char *argv[])
{
    int input_size0 = N * C * (NH + KH - 1) * (NW + KW - 1);
    int output_size;
   if (P == std::string("VALID")){
       output_size = N * C * ((NH - KH + 1) / S_height + 1) * ((NW - KW + 1) / S_width + 1);
   } else if (P == std::string("SAME")){
       output_size = N * C * (NH / S_height + 1) * (NW / S_width + 1);
   }

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size0 * sizeof(float));

    hipMalloc((void **)&Ad, input_size0 * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size0 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(21504, 1, 1);
    dim3 block(512, 1, 1);
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Cd, (float*)Ad);
        hipDeviceSynchronize();
    }
}
