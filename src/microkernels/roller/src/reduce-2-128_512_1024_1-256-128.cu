#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float A_shared[8192];
  float A_shared_local[2];
  compute_local[0] = 0.000000e+00f;
  compute_local[1] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31))];
    A_shared[(((int)threadIdx.x) + 128)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    A_shared[(((int)threadIdx.x) + 256)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    A_shared[(((int)threadIdx.x) + 384)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    A_shared[(((int)threadIdx.x) + 512)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
    A_shared[(((int)threadIdx.x) + 640)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
    A_shared[(((int)threadIdx.x) + 768)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
    A_shared[(((int)threadIdx.x) + 896)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 32768)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 36864)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 40960)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 45056)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 49152)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 53248)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 57344)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 61440)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 65536)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 69632)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 73728)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 77824)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 81920)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 86016)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 90112)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 94208)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 98304)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 102400)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 106496)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 110592)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 114688)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 118784)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 122880)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 126976)];
    A_shared[(((int)threadIdx.x) + 4096)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 131072)];
    A_shared[(((int)threadIdx.x) + 4224)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 135168)];
    A_shared[(((int)threadIdx.x) + 4352)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 139264)];
    A_shared[(((int)threadIdx.x) + 4480)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 143360)];
    A_shared[(((int)threadIdx.x) + 4608)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 147456)];
    A_shared[(((int)threadIdx.x) + 4736)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 151552)];
    A_shared[(((int)threadIdx.x) + 4864)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 155648)];
    A_shared[(((int)threadIdx.x) + 4992)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 159744)];
    A_shared[(((int)threadIdx.x) + 5120)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 163840)];
    A_shared[(((int)threadIdx.x) + 5248)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 167936)];
    A_shared[(((int)threadIdx.x) + 5376)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 172032)];
    A_shared[(((int)threadIdx.x) + 5504)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 176128)];
    A_shared[(((int)threadIdx.x) + 5632)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 180224)];
    A_shared[(((int)threadIdx.x) + 5760)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 184320)];
    A_shared[(((int)threadIdx.x) + 5888)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 188416)];
    A_shared[(((int)threadIdx.x) + 6016)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 192512)];
    A_shared[(((int)threadIdx.x) + 6144)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 196608)];
    A_shared[(((int)threadIdx.x) + 6272)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 200704)];
    A_shared[(((int)threadIdx.x) + 6400)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 204800)];
    A_shared[(((int)threadIdx.x) + 6528)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 208896)];
    A_shared[(((int)threadIdx.x) + 6656)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 212992)];
    A_shared[(((int)threadIdx.x) + 6784)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 217088)];
    A_shared[(((int)threadIdx.x) + 6912)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 221184)];
    A_shared[(((int)threadIdx.x) + 7040)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 225280)];
    A_shared[(((int)threadIdx.x) + 7168)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 229376)];
    A_shared[(((int)threadIdx.x) + 7296)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 233472)];
    A_shared[(((int)threadIdx.x) + 7424)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 237568)];
    A_shared[(((int)threadIdx.x) + 7552)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 241664)];
    A_shared[(((int)threadIdx.x) + 7680)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 245760)];
    A_shared[(((int)threadIdx.x) + 7808)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 249856)];
    A_shared[(((int)threadIdx.x) + 7936)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 253952)];
    A_shared[(((int)threadIdx.x) + 8064)] = A[(((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer * 32)) + (((int)threadIdx.x) & 31)) + 258048)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 32; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 32) + k_inner_outer)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.x) * 32) + k_inner_outer) + 4096)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
      compute_local[1] = (compute_local[1] + A_shared_local[1]);
    }
  }
  compute[((((int)blockIdx.x) * 256) + ((int)threadIdx.x))] = compute_local[0];
  compute[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) + 128)] = compute_local[1];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 256;
    int block_size = 128;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
