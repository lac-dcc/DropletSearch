#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float A_shared[4096];
  float A_shared_local[2];
  compute_local[0] = 0.000000e+00f;
  compute_local[1] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x))];
    A_shared[(((int)threadIdx.x) + 32)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 32)];
    A_shared[(((int)threadIdx.x) + 64)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 1024)];
    A_shared[(((int)threadIdx.x) + 96)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 1056)];
    A_shared[(((int)threadIdx.x) + 128)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 2048)];
    A_shared[(((int)threadIdx.x) + 160)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 2080)];
    A_shared[(((int)threadIdx.x) + 192)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 3072)];
    A_shared[(((int)threadIdx.x) + 224)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 3104)];
    A_shared[(((int)threadIdx.x) + 256)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 4096)];
    A_shared[(((int)threadIdx.x) + 288)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 4128)];
    A_shared[(((int)threadIdx.x) + 320)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 5120)];
    A_shared[(((int)threadIdx.x) + 352)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 5152)];
    A_shared[(((int)threadIdx.x) + 384)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 6144)];
    A_shared[(((int)threadIdx.x) + 416)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 6176)];
    A_shared[(((int)threadIdx.x) + 448)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 7168)];
    A_shared[(((int)threadIdx.x) + 480)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 7200)];
    A_shared[(((int)threadIdx.x) + 512)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 8192)];
    A_shared[(((int)threadIdx.x) + 544)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 8224)];
    A_shared[(((int)threadIdx.x) + 576)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 9216)];
    A_shared[(((int)threadIdx.x) + 608)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 9248)];
    A_shared[(((int)threadIdx.x) + 640)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 10240)];
    A_shared[(((int)threadIdx.x) + 672)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 10272)];
    A_shared[(((int)threadIdx.x) + 704)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 11264)];
    A_shared[(((int)threadIdx.x) + 736)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 11296)];
    A_shared[(((int)threadIdx.x) + 768)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 12288)];
    A_shared[(((int)threadIdx.x) + 800)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 12320)];
    A_shared[(((int)threadIdx.x) + 832)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 13312)];
    A_shared[(((int)threadIdx.x) + 864)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 13344)];
    A_shared[(((int)threadIdx.x) + 896)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 14336)];
    A_shared[(((int)threadIdx.x) + 928)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 14368)];
    A_shared[(((int)threadIdx.x) + 960)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 15360)];
    A_shared[(((int)threadIdx.x) + 992)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 15392)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 16384)];
    A_shared[(((int)threadIdx.x) + 1056)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 16416)];
    A_shared[(((int)threadIdx.x) + 1088)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 17408)];
    A_shared[(((int)threadIdx.x) + 1120)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 17440)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 18432)];
    A_shared[(((int)threadIdx.x) + 1184)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 18464)];
    A_shared[(((int)threadIdx.x) + 1216)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 19456)];
    A_shared[(((int)threadIdx.x) + 1248)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 19488)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 20480)];
    A_shared[(((int)threadIdx.x) + 1312)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 20512)];
    A_shared[(((int)threadIdx.x) + 1344)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 21504)];
    A_shared[(((int)threadIdx.x) + 1376)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 21536)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 22528)];
    A_shared[(((int)threadIdx.x) + 1440)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 22560)];
    A_shared[(((int)threadIdx.x) + 1472)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 23552)];
    A_shared[(((int)threadIdx.x) + 1504)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 23584)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 24576)];
    A_shared[(((int)threadIdx.x) + 1568)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 24608)];
    A_shared[(((int)threadIdx.x) + 1600)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 25600)];
    A_shared[(((int)threadIdx.x) + 1632)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 25632)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 26624)];
    A_shared[(((int)threadIdx.x) + 1696)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 26656)];
    A_shared[(((int)threadIdx.x) + 1728)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 27648)];
    A_shared[(((int)threadIdx.x) + 1760)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 27680)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 28672)];
    A_shared[(((int)threadIdx.x) + 1824)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 28704)];
    A_shared[(((int)threadIdx.x) + 1856)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 29696)];
    A_shared[(((int)threadIdx.x) + 1888)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 29728)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 30720)];
    A_shared[(((int)threadIdx.x) + 1952)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 30752)];
    A_shared[(((int)threadIdx.x) + 1984)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 31744)];
    A_shared[(((int)threadIdx.x) + 2016)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 31776)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 32768)];
    A_shared[(((int)threadIdx.x) + 2080)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 32800)];
    A_shared[(((int)threadIdx.x) + 2112)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 33792)];
    A_shared[(((int)threadIdx.x) + 2144)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 33824)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 34816)];
    A_shared[(((int)threadIdx.x) + 2208)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 34848)];
    A_shared[(((int)threadIdx.x) + 2240)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 35840)];
    A_shared[(((int)threadIdx.x) + 2272)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 35872)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 36864)];
    A_shared[(((int)threadIdx.x) + 2336)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 36896)];
    A_shared[(((int)threadIdx.x) + 2368)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 37888)];
    A_shared[(((int)threadIdx.x) + 2400)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 37920)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 38912)];
    A_shared[(((int)threadIdx.x) + 2464)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 38944)];
    A_shared[(((int)threadIdx.x) + 2496)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 39936)];
    A_shared[(((int)threadIdx.x) + 2528)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 39968)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 40960)];
    A_shared[(((int)threadIdx.x) + 2592)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 40992)];
    A_shared[(((int)threadIdx.x) + 2624)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 41984)];
    A_shared[(((int)threadIdx.x) + 2656)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 42016)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 43008)];
    A_shared[(((int)threadIdx.x) + 2720)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 43040)];
    A_shared[(((int)threadIdx.x) + 2752)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 44032)];
    A_shared[(((int)threadIdx.x) + 2784)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 44064)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 45056)];
    A_shared[(((int)threadIdx.x) + 2848)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 45088)];
    A_shared[(((int)threadIdx.x) + 2880)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 46080)];
    A_shared[(((int)threadIdx.x) + 2912)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 46112)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 47104)];
    A_shared[(((int)threadIdx.x) + 2976)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 47136)];
    A_shared[(((int)threadIdx.x) + 3008)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 48128)];
    A_shared[(((int)threadIdx.x) + 3040)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 48160)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 49152)];
    A_shared[(((int)threadIdx.x) + 3104)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 49184)];
    A_shared[(((int)threadIdx.x) + 3136)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 50176)];
    A_shared[(((int)threadIdx.x) + 3168)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 50208)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 51200)];
    A_shared[(((int)threadIdx.x) + 3232)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 51232)];
    A_shared[(((int)threadIdx.x) + 3264)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 52224)];
    A_shared[(((int)threadIdx.x) + 3296)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 52256)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 53248)];
    A_shared[(((int)threadIdx.x) + 3360)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 53280)];
    A_shared[(((int)threadIdx.x) + 3392)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 54272)];
    A_shared[(((int)threadIdx.x) + 3424)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 54304)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 55296)];
    A_shared[(((int)threadIdx.x) + 3488)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 55328)];
    A_shared[(((int)threadIdx.x) + 3520)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 56320)];
    A_shared[(((int)threadIdx.x) + 3552)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 56352)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 57344)];
    A_shared[(((int)threadIdx.x) + 3616)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 57376)];
    A_shared[(((int)threadIdx.x) + 3648)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 58368)];
    A_shared[(((int)threadIdx.x) + 3680)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 58400)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 59392)];
    A_shared[(((int)threadIdx.x) + 3744)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 59424)];
    A_shared[(((int)threadIdx.x) + 3776)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 60416)];
    A_shared[(((int)threadIdx.x) + 3808)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 60448)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 61440)];
    A_shared[(((int)threadIdx.x) + 3872)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 61472)];
    A_shared[(((int)threadIdx.x) + 3904)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 62464)];
    A_shared[(((int)threadIdx.x) + 3936)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 62496)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 63488)];
    A_shared[(((int)threadIdx.x) + 4000)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 63520)];
    A_shared[(((int)threadIdx.x) + 4032)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 64512)];
    A_shared[(((int)threadIdx.x) + 4064)] = A[((((((int)blockIdx.x) * 65536) + (k_outer * 64)) + ((int)threadIdx.x)) + 64544)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 64; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 64) + k_inner_outer)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.x) * 64) + k_inner_outer) + 2048)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
      compute_local[1] = (compute_local[1] + A_shared_local[1]);
    }
  }
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = compute_local[0];
  compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) + 32)] = compute_local[1];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 1024;
    int block_size = 32;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
