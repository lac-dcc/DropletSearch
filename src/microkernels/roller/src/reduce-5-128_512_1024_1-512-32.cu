#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

int M = 65536, N = 1024;


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float A_shared[8192];
  float A_shared_local[4];
  compute_local[0] = 0.000000e+00f;
  compute_local[1] = 0.000000e+00f;
  compute_local[2] = 0.000000e+00f;
  compute_local[3] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x))];
    A_shared[(((int)threadIdx.x) + 32)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 32)];
    A_shared[(((int)threadIdx.x) + 64)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 1024)];
    A_shared[(((int)threadIdx.x) + 96)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 1056)];
    A_shared[(((int)threadIdx.x) + 128)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 2048)];
    A_shared[(((int)threadIdx.x) + 160)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 2080)];
    A_shared[(((int)threadIdx.x) + 192)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 3072)];
    A_shared[(((int)threadIdx.x) + 224)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 3104)];
    A_shared[(((int)threadIdx.x) + 256)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 4096)];
    A_shared[(((int)threadIdx.x) + 288)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 4128)];
    A_shared[(((int)threadIdx.x) + 320)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 5120)];
    A_shared[(((int)threadIdx.x) + 352)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 5152)];
    A_shared[(((int)threadIdx.x) + 384)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 6144)];
    A_shared[(((int)threadIdx.x) + 416)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 6176)];
    A_shared[(((int)threadIdx.x) + 448)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 7168)];
    A_shared[(((int)threadIdx.x) + 480)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 7200)];
    A_shared[(((int)threadIdx.x) + 512)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 8192)];
    A_shared[(((int)threadIdx.x) + 544)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 8224)];
    A_shared[(((int)threadIdx.x) + 576)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 9216)];
    A_shared[(((int)threadIdx.x) + 608)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 9248)];
    A_shared[(((int)threadIdx.x) + 640)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 10240)];
    A_shared[(((int)threadIdx.x) + 672)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 10272)];
    A_shared[(((int)threadIdx.x) + 704)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 11264)];
    A_shared[(((int)threadIdx.x) + 736)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 11296)];
    A_shared[(((int)threadIdx.x) + 768)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 12288)];
    A_shared[(((int)threadIdx.x) + 800)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 12320)];
    A_shared[(((int)threadIdx.x) + 832)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 13312)];
    A_shared[(((int)threadIdx.x) + 864)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 13344)];
    A_shared[(((int)threadIdx.x) + 896)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 14336)];
    A_shared[(((int)threadIdx.x) + 928)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 14368)];
    A_shared[(((int)threadIdx.x) + 960)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 15360)];
    A_shared[(((int)threadIdx.x) + 992)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 15392)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 16384)];
    A_shared[(((int)threadIdx.x) + 1056)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 16416)];
    A_shared[(((int)threadIdx.x) + 1088)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 17408)];
    A_shared[(((int)threadIdx.x) + 1120)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 17440)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 18432)];
    A_shared[(((int)threadIdx.x) + 1184)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 18464)];
    A_shared[(((int)threadIdx.x) + 1216)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 19456)];
    A_shared[(((int)threadIdx.x) + 1248)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 19488)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 20480)];
    A_shared[(((int)threadIdx.x) + 1312)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 20512)];
    A_shared[(((int)threadIdx.x) + 1344)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 21504)];
    A_shared[(((int)threadIdx.x) + 1376)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 21536)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 22528)];
    A_shared[(((int)threadIdx.x) + 1440)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 22560)];
    A_shared[(((int)threadIdx.x) + 1472)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 23552)];
    A_shared[(((int)threadIdx.x) + 1504)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 23584)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 24576)];
    A_shared[(((int)threadIdx.x) + 1568)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 24608)];
    A_shared[(((int)threadIdx.x) + 1600)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 25600)];
    A_shared[(((int)threadIdx.x) + 1632)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 25632)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 26624)];
    A_shared[(((int)threadIdx.x) + 1696)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 26656)];
    A_shared[(((int)threadIdx.x) + 1728)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 27648)];
    A_shared[(((int)threadIdx.x) + 1760)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 27680)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 28672)];
    A_shared[(((int)threadIdx.x) + 1824)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 28704)];
    A_shared[(((int)threadIdx.x) + 1856)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 29696)];
    A_shared[(((int)threadIdx.x) + 1888)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 29728)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 30720)];
    A_shared[(((int)threadIdx.x) + 1952)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 30752)];
    A_shared[(((int)threadIdx.x) + 1984)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 31744)];
    A_shared[(((int)threadIdx.x) + 2016)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 31776)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 32768)];
    A_shared[(((int)threadIdx.x) + 2080)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 32800)];
    A_shared[(((int)threadIdx.x) + 2112)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 33792)];
    A_shared[(((int)threadIdx.x) + 2144)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 33824)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 34816)];
    A_shared[(((int)threadIdx.x) + 2208)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 34848)];
    A_shared[(((int)threadIdx.x) + 2240)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 35840)];
    A_shared[(((int)threadIdx.x) + 2272)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 35872)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 36864)];
    A_shared[(((int)threadIdx.x) + 2336)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 36896)];
    A_shared[(((int)threadIdx.x) + 2368)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 37888)];
    A_shared[(((int)threadIdx.x) + 2400)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 37920)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 38912)];
    A_shared[(((int)threadIdx.x) + 2464)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 38944)];
    A_shared[(((int)threadIdx.x) + 2496)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 39936)];
    A_shared[(((int)threadIdx.x) + 2528)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 39968)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 40960)];
    A_shared[(((int)threadIdx.x) + 2592)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 40992)];
    A_shared[(((int)threadIdx.x) + 2624)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 41984)];
    A_shared[(((int)threadIdx.x) + 2656)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 42016)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 43008)];
    A_shared[(((int)threadIdx.x) + 2720)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 43040)];
    A_shared[(((int)threadIdx.x) + 2752)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 44032)];
    A_shared[(((int)threadIdx.x) + 2784)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 44064)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 45056)];
    A_shared[(((int)threadIdx.x) + 2848)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 45088)];
    A_shared[(((int)threadIdx.x) + 2880)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 46080)];
    A_shared[(((int)threadIdx.x) + 2912)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 46112)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 47104)];
    A_shared[(((int)threadIdx.x) + 2976)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 47136)];
    A_shared[(((int)threadIdx.x) + 3008)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 48128)];
    A_shared[(((int)threadIdx.x) + 3040)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 48160)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 49152)];
    A_shared[(((int)threadIdx.x) + 3104)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 49184)];
    A_shared[(((int)threadIdx.x) + 3136)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 50176)];
    A_shared[(((int)threadIdx.x) + 3168)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 50208)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 51200)];
    A_shared[(((int)threadIdx.x) + 3232)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 51232)];
    A_shared[(((int)threadIdx.x) + 3264)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 52224)];
    A_shared[(((int)threadIdx.x) + 3296)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 52256)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 53248)];
    A_shared[(((int)threadIdx.x) + 3360)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 53280)];
    A_shared[(((int)threadIdx.x) + 3392)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 54272)];
    A_shared[(((int)threadIdx.x) + 3424)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 54304)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 55296)];
    A_shared[(((int)threadIdx.x) + 3488)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 55328)];
    A_shared[(((int)threadIdx.x) + 3520)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 56320)];
    A_shared[(((int)threadIdx.x) + 3552)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 56352)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 57344)];
    A_shared[(((int)threadIdx.x) + 3616)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 57376)];
    A_shared[(((int)threadIdx.x) + 3648)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 58368)];
    A_shared[(((int)threadIdx.x) + 3680)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 58400)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 59392)];
    A_shared[(((int)threadIdx.x) + 3744)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 59424)];
    A_shared[(((int)threadIdx.x) + 3776)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 60416)];
    A_shared[(((int)threadIdx.x) + 3808)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 60448)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 61440)];
    A_shared[(((int)threadIdx.x) + 3872)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 61472)];
    A_shared[(((int)threadIdx.x) + 3904)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 62464)];
    A_shared[(((int)threadIdx.x) + 3936)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 62496)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 63488)];
    A_shared[(((int)threadIdx.x) + 4000)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 63520)];
    A_shared[(((int)threadIdx.x) + 4032)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 64512)];
    A_shared[(((int)threadIdx.x) + 4064)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 64544)];
    A_shared[(((int)threadIdx.x) + 4096)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 65536)];
    A_shared[(((int)threadIdx.x) + 4128)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 65568)];
    A_shared[(((int)threadIdx.x) + 4160)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 66560)];
    A_shared[(((int)threadIdx.x) + 4192)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 66592)];
    A_shared[(((int)threadIdx.x) + 4224)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 67584)];
    A_shared[(((int)threadIdx.x) + 4256)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 67616)];
    A_shared[(((int)threadIdx.x) + 4288)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 68608)];
    A_shared[(((int)threadIdx.x) + 4320)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 68640)];
    A_shared[(((int)threadIdx.x) + 4352)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 69632)];
    A_shared[(((int)threadIdx.x) + 4384)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 69664)];
    A_shared[(((int)threadIdx.x) + 4416)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 70656)];
    A_shared[(((int)threadIdx.x) + 4448)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 70688)];
    A_shared[(((int)threadIdx.x) + 4480)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 71680)];
    A_shared[(((int)threadIdx.x) + 4512)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 71712)];
    A_shared[(((int)threadIdx.x) + 4544)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 72704)];
    A_shared[(((int)threadIdx.x) + 4576)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 72736)];
    A_shared[(((int)threadIdx.x) + 4608)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 73728)];
    A_shared[(((int)threadIdx.x) + 4640)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 73760)];
    A_shared[(((int)threadIdx.x) + 4672)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 74752)];
    A_shared[(((int)threadIdx.x) + 4704)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 74784)];
    A_shared[(((int)threadIdx.x) + 4736)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 75776)];
    A_shared[(((int)threadIdx.x) + 4768)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 75808)];
    A_shared[(((int)threadIdx.x) + 4800)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 76800)];
    A_shared[(((int)threadIdx.x) + 4832)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 76832)];
    A_shared[(((int)threadIdx.x) + 4864)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 77824)];
    A_shared[(((int)threadIdx.x) + 4896)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 77856)];
    A_shared[(((int)threadIdx.x) + 4928)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 78848)];
    A_shared[(((int)threadIdx.x) + 4960)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 78880)];
    A_shared[(((int)threadIdx.x) + 4992)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 79872)];
    A_shared[(((int)threadIdx.x) + 5024)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 79904)];
    A_shared[(((int)threadIdx.x) + 5056)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 80896)];
    A_shared[(((int)threadIdx.x) + 5088)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 80928)];
    A_shared[(((int)threadIdx.x) + 5120)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 81920)];
    A_shared[(((int)threadIdx.x) + 5152)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 81952)];
    A_shared[(((int)threadIdx.x) + 5184)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 82944)];
    A_shared[(((int)threadIdx.x) + 5216)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 82976)];
    A_shared[(((int)threadIdx.x) + 5248)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 83968)];
    A_shared[(((int)threadIdx.x) + 5280)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 84000)];
    A_shared[(((int)threadIdx.x) + 5312)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 84992)];
    A_shared[(((int)threadIdx.x) + 5344)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 85024)];
    A_shared[(((int)threadIdx.x) + 5376)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 86016)];
    A_shared[(((int)threadIdx.x) + 5408)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 86048)];
    A_shared[(((int)threadIdx.x) + 5440)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 87040)];
    A_shared[(((int)threadIdx.x) + 5472)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 87072)];
    A_shared[(((int)threadIdx.x) + 5504)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 88064)];
    A_shared[(((int)threadIdx.x) + 5536)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 88096)];
    A_shared[(((int)threadIdx.x) + 5568)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 89088)];
    A_shared[(((int)threadIdx.x) + 5600)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 89120)];
    A_shared[(((int)threadIdx.x) + 5632)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 90112)];
    A_shared[(((int)threadIdx.x) + 5664)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 90144)];
    A_shared[(((int)threadIdx.x) + 5696)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 91136)];
    A_shared[(((int)threadIdx.x) + 5728)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 91168)];
    A_shared[(((int)threadIdx.x) + 5760)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 92160)];
    A_shared[(((int)threadIdx.x) + 5792)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 92192)];
    A_shared[(((int)threadIdx.x) + 5824)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 93184)];
    A_shared[(((int)threadIdx.x) + 5856)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 93216)];
    A_shared[(((int)threadIdx.x) + 5888)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 94208)];
    A_shared[(((int)threadIdx.x) + 5920)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 94240)];
    A_shared[(((int)threadIdx.x) + 5952)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 95232)];
    A_shared[(((int)threadIdx.x) + 5984)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 95264)];
    A_shared[(((int)threadIdx.x) + 6016)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 96256)];
    A_shared[(((int)threadIdx.x) + 6048)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 96288)];
    A_shared[(((int)threadIdx.x) + 6080)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 97280)];
    A_shared[(((int)threadIdx.x) + 6112)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 97312)];
    A_shared[(((int)threadIdx.x) + 6144)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 98304)];
    A_shared[(((int)threadIdx.x) + 6176)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 98336)];
    A_shared[(((int)threadIdx.x) + 6208)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 99328)];
    A_shared[(((int)threadIdx.x) + 6240)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 99360)];
    A_shared[(((int)threadIdx.x) + 6272)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 100352)];
    A_shared[(((int)threadIdx.x) + 6304)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 100384)];
    A_shared[(((int)threadIdx.x) + 6336)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 101376)];
    A_shared[(((int)threadIdx.x) + 6368)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 101408)];
    A_shared[(((int)threadIdx.x) + 6400)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 102400)];
    A_shared[(((int)threadIdx.x) + 6432)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 102432)];
    A_shared[(((int)threadIdx.x) + 6464)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 103424)];
    A_shared[(((int)threadIdx.x) + 6496)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 103456)];
    A_shared[(((int)threadIdx.x) + 6528)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 104448)];
    A_shared[(((int)threadIdx.x) + 6560)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 104480)];
    A_shared[(((int)threadIdx.x) + 6592)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 105472)];
    A_shared[(((int)threadIdx.x) + 6624)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 105504)];
    A_shared[(((int)threadIdx.x) + 6656)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 106496)];
    A_shared[(((int)threadIdx.x) + 6688)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 106528)];
    A_shared[(((int)threadIdx.x) + 6720)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 107520)];
    A_shared[(((int)threadIdx.x) + 6752)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 107552)];
    A_shared[(((int)threadIdx.x) + 6784)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 108544)];
    A_shared[(((int)threadIdx.x) + 6816)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 108576)];
    A_shared[(((int)threadIdx.x) + 6848)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 109568)];
    A_shared[(((int)threadIdx.x) + 6880)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 109600)];
    A_shared[(((int)threadIdx.x) + 6912)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 110592)];
    A_shared[(((int)threadIdx.x) + 6944)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 110624)];
    A_shared[(((int)threadIdx.x) + 6976)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 111616)];
    A_shared[(((int)threadIdx.x) + 7008)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 111648)];
    A_shared[(((int)threadIdx.x) + 7040)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 112640)];
    A_shared[(((int)threadIdx.x) + 7072)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 112672)];
    A_shared[(((int)threadIdx.x) + 7104)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 113664)];
    A_shared[(((int)threadIdx.x) + 7136)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 113696)];
    A_shared[(((int)threadIdx.x) + 7168)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 114688)];
    A_shared[(((int)threadIdx.x) + 7200)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 114720)];
    A_shared[(((int)threadIdx.x) + 7232)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 115712)];
    A_shared[(((int)threadIdx.x) + 7264)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 115744)];
    A_shared[(((int)threadIdx.x) + 7296)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 116736)];
    A_shared[(((int)threadIdx.x) + 7328)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 116768)];
    A_shared[(((int)threadIdx.x) + 7360)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 117760)];
    A_shared[(((int)threadIdx.x) + 7392)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 117792)];
    A_shared[(((int)threadIdx.x) + 7424)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 118784)];
    A_shared[(((int)threadIdx.x) + 7456)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 118816)];
    A_shared[(((int)threadIdx.x) + 7488)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 119808)];
    A_shared[(((int)threadIdx.x) + 7520)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 119840)];
    A_shared[(((int)threadIdx.x) + 7552)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 120832)];
    A_shared[(((int)threadIdx.x) + 7584)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 120864)];
    A_shared[(((int)threadIdx.x) + 7616)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 121856)];
    A_shared[(((int)threadIdx.x) + 7648)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 121888)];
    A_shared[(((int)threadIdx.x) + 7680)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 122880)];
    A_shared[(((int)threadIdx.x) + 7712)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 122912)];
    A_shared[(((int)threadIdx.x) + 7744)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 123904)];
    A_shared[(((int)threadIdx.x) + 7776)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 123936)];
    A_shared[(((int)threadIdx.x) + 7808)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 124928)];
    A_shared[(((int)threadIdx.x) + 7840)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 124960)];
    A_shared[(((int)threadIdx.x) + 7872)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 125952)];
    A_shared[(((int)threadIdx.x) + 7904)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 125984)];
    A_shared[(((int)threadIdx.x) + 7936)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 126976)];
    A_shared[(((int)threadIdx.x) + 7968)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 127008)];
    A_shared[(((int)threadIdx.x) + 8000)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 128000)];
    A_shared[(((int)threadIdx.x) + 8032)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 128032)];
    A_shared[(((int)threadIdx.x) + 8064)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 129024)];
    A_shared[(((int)threadIdx.x) + 8096)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 129056)];
    A_shared[(((int)threadIdx.x) + 8128)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 130048)];
    A_shared[(((int)threadIdx.x) + 8160)] = A[((((((int)blockIdx.x) * 131072) + (k_outer * 64)) + ((int)threadIdx.x)) + 130080)];
    __syncthreads();
    for (int k_inner_outer = 0; k_inner_outer < 64; ++k_inner_outer) {
      A_shared_local[0] = A_shared[((((int)threadIdx.x) * 64) + k_inner_outer)];
      A_shared_local[1] = A_shared[(((((int)threadIdx.x) * 64) + k_inner_outer) + 2048)];
      A_shared_local[2] = A_shared[(((((int)threadIdx.x) * 64) + k_inner_outer) + 4096)];
      A_shared_local[3] = A_shared[(((((int)threadIdx.x) * 64) + k_inner_outer) + 6144)];
      compute_local[0] = (compute_local[0] + A_shared_local[0]);
      compute_local[1] = (compute_local[1] + A_shared_local[1]);
      compute_local[2] = (compute_local[2] + A_shared_local[2]);
      compute_local[3] = (compute_local[3] + A_shared_local[3]);
    }
  }
  compute[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = compute_local[0];
  compute[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 32)] = compute_local[1];
  compute[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 64)] = compute_local[2];
  compute[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 96)] = compute_local[3];
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size = M * N;
    int output_size = M;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *Ah, *Ch;
    float *Ad, *Cd;
    Ah = (float*)malloc(input_size * sizeof(float));
    Ch = (float*)malloc(output_size * sizeof(float));

    hipMalloc((void **)&Ad, input_size * sizeof(float));
    hipMalloc((void **)&Cd, output_size * sizeof(float));

    srand(1);
    for (int i = 0; i < input_size; ++ i)
        Ah[i] = 1;

    hipMemcpy(Ad, Ah, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, output_size * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = 512;
    int block_size = 32;
    dim3 grid(grid_size, 1, 1);
    dim3 block(block_size, 1, 1);

    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, default_function_kernel0, block_size, 0);
    fprintf(stderr, "Active blocks per SM = %d\n", numBlocks);
 
    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Cd);
        hipDeviceSynchronize();
    }
}
