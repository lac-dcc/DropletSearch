#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>
#include "conv.cuh"

int N = 64, F = 64;
int C = 3, P = 0, S = 2;
int NH = 230, KH = 7;
int NW = 230, KW = 7;

extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute);

int main(int argc, char *argv[])
{
        std::string path;
        for (int i = 1; i < argc; i++) {
                printf("%d: %s\n", i, argv[i]);
                if (strcmp(argv[i], "-n" ) == 0) N = atoi(argv[i+1]);
                if (strcmp(argv[i], "-f" ) == 0) F = atoi(argv[i+1]);
                if (strcmp(argv[i], "-C" ) == 0) C = atoi(argv[i+1]);
                if (strcmp(argv[i], "-nh") == 0) NH = atoi(argv[i+1]);
                if (strcmp(argv[i], "-nw") == 0) NW = atoi(argv[i+1]);
                if (strcmp(argv[i], "-kh") == 0) KH = atoi(argv[i+1]);
                if (strcmp(argv[i], "-kw") == 0) KW = atoi(argv[i+1]);
                if (strcmp(argv[i], "-s") == 0)  S = atoi(argv[i+1]);
                if (strcmp(argv[i], "-pad") == 0) P = atoi(argv[i+1]);
                if (strcmp(argv[i], "-p" ) == 0) path = argv[i+1];
        }
        std::string code_path = path + "/my_kernel.cc";
        std::string mod_path = path + "/my_kernel.out";
        int input_size0 = N * C * NH * NW;
        int input_size1 = F * C * KH * KW;
        int output_size = N * F * ((NH - KH + 2 * P) / S + 1) * ((NW - KW + 2 * P) / S + 1);
        printf("n: %d c: %d f: %d nh: %d nh: %d kw: %d kw: %d p: %d s: %d\npath: %s code_path: %s mod_path: %s\n",
                N, C, F, NH, NH, KW, KW, P, S, path.c_str(), code_path.c_str(), mod_path.c_str());

        checkCudaErrors(hipInit(0));
        hipDevice_t device;
        checkCudaErrors(hipDeviceGet(&device, 0));
        hipCtx_t context;
        checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

        hipModule_t cuModule_;
        hipFunction_t cuda_func_;

        float *Ah, *Bh, *Ch;
        hipDeviceptr_t Ad, Bd, Cd;
        Ah = (float*)malloc(input_size0 * sizeof(float));
        Bh = (float*)malloc(input_size1 * sizeof(float));
        Ch = (float*)malloc(output_size * sizeof(float));

        // hipMalloc((void **)&Ad, input_size0 * sizeof(float));
        // hipMalloc((void **)&Bd, input_size1 * sizeof(float));
        // hipMalloc((void **)&Cd, output_size * sizeof(float));

        checkCudaErrors(hipMalloc(&Ad, sizeof(float) * input_size0));
        checkCudaErrors(hipMalloc(&Bd, sizeof(float) * input_size1));
        checkCudaErrors(hipMalloc(&Cd, sizeof(float) * output_size));

        void* param[] = {&Ad, &Bd, &Cd};

        srand(1);
        for (int i = 0; i < input_size0; ++ i) Ah[i] = rand();
        for (int i = 0; i < input_size1; ++ i) Bh[i] = rand();

        checkCudaErrors(hipMemcpyHtoD(Ad, Ah, input_size0 * sizeof(float)));
        checkCudaErrors(hipMemcpyHtoD(Bd, Bh, input_size1 * sizeof(float)));

        // checkCudaErrors(hipModuleLoad(&cuModule_, mod_path.c_str()));
        // checkCudaErrors(hipModuleGetFunction(&cuda_func_, cuModule_, "template_op_kernel0"));
        // FILE* fp = fopen(code_path.c_str(), "r");
        // int block_x = 1, block_y = 1, thread_x = 1, thread_y = 1;
        // while (!feof(fp))
        // {
        //         char *line;
        //         line = (char*)malloc(2000 * sizeof(char));
        //         fgets(line, 2000, fp);
        //         std::string std_line = std::string(line);
        //         if (int(std_line.find("[thread_extent] blockIdx.x")) > -1)
        //         {
        //                 int k = std_line.rfind("=");
        //                 block_x = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
        //         }
        //         if (int(std_line.find("[thread_extent] blockIdx.y")) > -1)
        //         {
        //                         int k = std_line.rfind("=");
        //                 block_y = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
        //         }
        //         if (int(std_line.find("[thread_extent] threadIdx.x")) > -1)
        //         {
        //                 int k = std_line.rfind("=");
        //                 thread_x = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
        //         }
        //         if (int(std_line.find("[thread_extent] threadIdx.y")) > -1)
        //         {
        //                 int k = std_line.rfind("=");
        //                 thread_y = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
        //         }
        // }
        // printf("path: %s Grid: (%d %d, 1) Block: (%d %d 1)\n", path.c_str(), block_x, block_y, thread_x, thread_y);
        // for (int i = 0; i < 1; ++ i)
        // {
        //         checkCudaErrors(hipModuleLaunchKernel(cuda_func_, block_x, block_y, 1, thread_x, thread_y, 1, 0, 0, (void**) param, 0));
        //         hipDeviceSynchronize();
        // }

        dim3 grid(4096, 1, 1);
        dim3 block(224, 1, 1);
        for (int i = 0; i < 1; ++i)
        {
                int r = hipFuncSetAttribute(reinterpret_cast<const void*>(default_function_kernel0), hipFuncAttributeMaxDynamicSharedMemorySize, 96 * 1024);
                printf("hipFuncSetAttribute: %d\n", r);
                default_function_kernel0<<<grid, block>>>((float*)Ad, (float*)Bd, (float*)Cd);
                r = hipDeviceSynchronize();
                printf("hipDeviceSynchronize: %d\n", r);
        }
}